#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <chrono>

#define N 10e4 // Size of vectors
#define ceil(x,y) (((x)+(y)-1)/(y)) 
#define nShareMem 256

// Kernel function to compute the inner product of two vectors on GPU
__global__ void innerProductKernel(const float* A, const float* B, double* partialSums) {
    __shared__ double temp[nShareMem];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    temp[threadIdx.x] = A[idx] * B[idx];
    __syncthreads();

    // Reduce within block
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            temp[threadIdx.x] += temp[threadIdx.x + i];
        }
        __syncthreads();
    }
    
    // Store partial sum for block
    if (threadIdx.x == 0) {
        partialSums[blockIdx.x] = temp[0];
    }
}

float innerProductCUDA(const float* A, const float* B) {
    float* d_A;
    float* d_B;
    double* d_partialSums;

    double *partialSums = new double [ceil(N, nShareMem)];

    // float partialSums[N / nShareMem + 1] = {0};
    double h_C = 0;

    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_partialSums, ceil(N, nShareMem) * sizeof(double));

    dim3 dimBlock(nShareMem, 1, 1);
    dim3 dimGrid(ceil(N, dimBlock.x), 1, 1);
    
    auto start_cuda = std::chrono::high_resolution_clock::now();



    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    innerProductKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_partialSums);

    hipMemcpy(partialSums, d_partialSums, ceil(N, nShareMem) * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < ceil(N, dimBlock.x); i++) {
        h_C += partialSums[i];
    }
    auto stop_cuda = std::chrono::high_resolution_clock::now();
    auto duration_cuda = std::chrono::duration_cast<std::chrono::microseconds>(stop_cuda - start_cuda);
    std::cout << "CUDA_REAL: " << duration_cuda.count() <<" microseconds"<<std::endl;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_partialSums);

    return h_C;
}





float cublas_dot_product(float* x, float* y, int Narr)
{
    // Initialize cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate memory on the device for the vectors
    float* d_x, * d_y;
    hipMalloc(&d_x, Narr * sizeof(float));
    hipMalloc(&d_y, Narr * sizeof(float));


    auto start_cublas = std::chrono::high_resolution_clock::now();
    

    // Copy the vectors from host to device
    hipMemcpy(d_x, x, Narr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, Narr * sizeof(float), hipMemcpyHostToDevice);

    // Calculate the inner product of the vectors using cublas
    float result;
    hipblasSdot(handle, Narr, d_x, 1, d_y, 1, &result);
    auto stop_cublas = std::chrono::high_resolution_clock::now();
    auto duration_cublas = std::chrono::duration_cast<std::chrono::microseconds>(stop_cublas - start_cublas);
    std::cout << "CUBLAS: " <<duration_cublas.count()<<" microseconds\n";
    // Free memory on the device
    hipFree(d_x);
    hipFree(d_y);

    // Destroy cublas handle
    hipblasDestroy(handle);

    return result;
}


// Function to compute the inner product of two vectors using the CPU
double innerProductCPU(const float* A, const float* B) {
    double result = 0;
    for (int i = 0; i < N; i++) {
        result += A[i] * B[i];
    }
    return result;
}

int main() {
    std::cout<<"array with length of "<<N<<std::endl;
    std::vector<float> A(N);
    std::vector<float> B(N);

    // Initialize vectors with random values
    for (int i = 0; i < N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // Compute inner product using CUDA

    float result_cuda = innerProductCUDA(A.data(), B.data());
    float result_cublas = cublas_dot_product(A.data(), B.data(),N);

    // Compute inner product using CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    float result_cpu = innerProductCPU(A.data(), B.data());
    auto stop_cpu = std::chrono::high_resolution_clock::now();
    auto duration_cpu = std::chrono::duration_cast<std::chrono::microseconds>(stop_cpu - start_cpu);

    

    
    
    // Print execution times
    // std::cout << "CUDA: " << duration_cuda.count() << " microseconds\n";
    
    
    std::cout << "CPU: " << duration_cpu.count() << " microseconds\n";  

    // Check if results are equal
    if (result_cpu == result_cuda && result_cpu == result_cublas) {
        std::cout << "Results are equal\n";
    } else {
        std::cout << "Results are not equal\n "<<std::endl;
        std::cout << "Result cpu=" <<result_cpu <<" Result cublas="<<result_cublas<< " Reulst cuda=" << result_cuda<<std::endl;
        
    }
    return 0;
}
